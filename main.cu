#include "hip/hip_runtime.h"
#include "main.h"

int main(){
  std::srand(std::time(0)); // set seed as current time;
  
  // LCG stuff
  std::cout << "Creating empty lcg..." << std::endl;
  Rng_Lcg l1 = Rng_Lcg();
  std::cout << "l1 seed = " << l1.Get_Seed() << std::endl;

  Seeds seed[1];
  std::cout << "Initializing seeds..." << std::endl;
  initialize(seed, 1);
  Rng_Lcg l2 = Rng_Lcg(seed[0].sa);

  std::cout << "0th seed: " << l2.Get_Seed() << std::endl;
  l2.Random();
  std::cout << "Lcg random number: " << l2.Get_Seed() << std::endl; 

  // Tausworthe stuff
  std::cout << "Creating empty Tausworthe..." << std::endl;
  Rng_Tausworthe t1 = Rng_Tausworthe();
  std::cout << "t1 seed = " << t1.Get_Seed() << std::endl;
  Tausworthe_Parameters par1;
  par1.k1 = 13; par1.k2 = 19; par1.k3 = 12; par1.m = 4294967294UL;
  Rng_Tausworthe t2 = Rng_Tausworthe(seed[0].sb, par1);
 
  std::cout << "0th seed: " << t2.Get_Seed() << std::endl;
  t2.Random();
  std::cout << "Tausworthe random number: " << t2.Get_Seed() << std::endl;

  // Combined stuff
  Tausworthe_Parameters par2;
  par2.k1 = 2; par2.k2 = 25; par2.k3 = 4; par2.m = 4294967288UL;
  Tausworthe_Parameters par3;
  par3.k1 = 3; par3.k2 = 11; par3.k3 = 17; par3.m = 4294967280UL;
  
  std::cout << "Creating empty combined..." << std::endl;
  Rng_Combined c = Rng_Combined();
  std::cout << "The standard CTOR 4 seeds are: " << (c.Get_Seeds()).sa << ", " << (c.Get_Seeds()).sb << ", " 
            << (c.Get_Seeds()).sc << ", " << (c.Get_Seeds()).sd << std::endl;
 
  Rng_Combined c1 = Rng_Combined(seed[0], par1, par2, par3);
  std::cout << "The new 4 seeds are: " << (c1.Get_Seeds()).sa << ", " << (c1.Get_Seeds()).sb << ", " 
            << (c1.Get_Seeds()).sc << ", " << (c1.Get_Seeds()).sd << std::endl;
  c1.Random(); // randomize element
  std::cout << "After randomizing, the 4 seeds are: " << (c1.Get_Seeds()).sa << ", " << (c1.Get_Seeds()).sb << ", " 
            << (c1.Get_Seeds()).sc << ", " << (c1.Get_Seeds()).sd << std::endl;
  
  std::cout << "Here is a random uniform number (µ = 0.5, σ² = 1/12): " << c1.Get_Rand() << std::endl;
  c1.RandomBoxMuller(); // create a pair of gaussian numbers  
  std::cout << "Here are 2 gaussian numbers (µ = 0, σ² = 1): "
            << "g1 = " << c1.Get_Gauss1() << ", g2 = " << c1.Get_Gauss2() << std::endl;

  // Do some statistics
  double mean = 0, variance = 0;
  double meanGauss1 = 0, varianceGauss1 = 0;
  double meanGauss2 = 0, varianceGauss2 = 0;

  Seeds ss[DIM];
  initialize(ss, DIM); 
  
  Rng_Combined cc[DIM];
  // uniform
  std::cout << std::endl << "Preparing an array with " << DIM << " elements..." << std::endl;
  for (int i=0; i<DIM; ++i){
    cc[i] = Rng_Combined(ss[i], par1, par2, par3); // initialize
    cc[i].Random();
    mean = (i*mean + cc[i].Get_Rand())/(i+1);
    variance = (i*variance + (cc[i].Get_Rand()-.5)*(cc[i].Get_Rand()-.5))/(i+1);
  }

  // gaussian
  for(int i=0; i<DIM; ++i){
    cc[i] = Rng_Combined(ss[i], par1, par2, par3); // re-initialize
    cc[i].RandomBoxMuller();
    meanGauss1 = (i*meanGauss1 + cc[i].Get_Gauss1())/(i+1);
    varianceGauss1 = (i*varianceGauss1 + (cc[i].Get_Gauss1()-0.)*(cc[i].Get_Gauss1()-0.))/(i+1);
    meanGauss2 = (i*meanGauss2 + cc[i].Get_Gauss2())/(i+1);
    varianceGauss2 = (i*varianceGauss2 + (cc[i].Get_Gauss2()-0.)*(cc[i].Get_Gauss2()-0.))/(i+1);
  }
  std::cout << "mean = " << mean << ", variance = " << variance 
            << ", relative error = " << std::fabs((mean-.5)/variance) << "(σ)" << std::endl;
  if (std::fabs((mean-.5)/variance)<1){
    std::cout << "Uniform generator passed the test!" << std::endl;
  }
  std::cout << "meanGauss1 = " << meanGauss1 << ", variance1 = " << varianceGauss1 
            <<", relative error = " << std::fabs((meanGauss1-0.)/varianceGauss1) << "(σ)" << std::endl;
  std::cout << "meanGauss2 = " << meanGauss2 << ", variance2 = " << varianceGauss2 
            <<", relative error = " << std::fabs((meanGauss2-0.)/varianceGauss2) << "(σ)" << std::endl;
  if (std::fabs((meanGauss1-0./varianceGauss1))<1 && std::fabs((meanGauss2-0./varianceGauss2))){
    std::cout << "Box Muller generator passed the test!" << std::endl;
  }

  return 0;
}
