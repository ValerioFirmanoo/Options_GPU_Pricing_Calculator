#include "hip/hip_runtime.h"
#include "stochastic_process.h"
#include <iostream>
#include <fstream> 
#include <ctime>
#include <iomanip>
#include "options/opt_european_call.h"
//#include "options/opt_european_put.h"
#include "forward_contract.h"
//#include "structs.h"
#include "main.h"

__global__ void pricing (Market_Data market_data, GPU_Parameters gpu_par,Tausworthe_Parameters par1,Tausworthe_Parameters par2,Tausworthe_Parameters par3,Seeds *d_seed, double *d_pay,double *d_stock){
 
//Rng_Combined d_omega[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];


  Rng_Combined *d_omega = new Rng_Combined[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];
  Stochastic_Process *d_SPX = new Stochastic_Process[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];
  Opt_European_Call *d_oec = new Opt_European_Call[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];

 int tid = threadIdx.x + blockIdx.x * blockDim.x;
 
 d_omega[tid] = Rng_Combined(d_seed[tid],par1,par2,par3);
 d_SPX[tid]= Stochastic_Process(&d_omega[tid]);
 d_SPX[tid].Create_Path_Exact(market_data.steps , market_data.S , market_data.T , market_data.sigma , market_data.r );
 d_oec[tid]= Opt_European_Call(&d_SPX[tid]);

//fino a qui tutto bene
 

//d_pay[tid]=d_oec[tid].Payoff(market_data.E , market_data.steps );
d_stock[tid]=d_SPX[tid].Get_path()[market_data.steps -1];  
d_pay[tid]=d_omega[tid].Get_Gauss();

//delete[] d_omega; delete[] d_SPX; delete[] d_oec;

/*
d_pay[tid]=12.3;
d_stock[tid]=0.4;
*/
}


int main(){
  std::srand(std::time(0));                        // set seed as current time;
  std::ofstream outpath_exact("path_exact.dat");   // output stream variable
  std::ofstream outpath_euler("path_euler.dat");   // output stream variable
  std::ofstream out_call("call_opt.dat");          // output stream variable
  std::cout << std::fixed << std::setprecision(2); // 2 decimal numbers

  Tausworthe_Parameters par1;
  Tausworthe_Parameters par2;
  Tausworthe_Parameters par3;
  par1.k1 = 13; par1.k2 = 19; par1.k3 = 12; par1.m = 4294967294UL;
  par2.k1 = 2; par2.k2 = 25; par2.k3 = 4; par2.m = 4294967288UL;
  par3.k1 = 3; par3.k2 = 11; par3.k3 = 17; par3.m = 4294967280UL;
//  Rng_Combined omega = Rng_Combined(seed[0], par1, par2, par3);

// setting gpu parameters
  GPU_Parameters gpu_par;
  gpu_par.THREADS_PER_BLOCK=32;
  gpu_par.BLOCK_PER_GRID=4;

//setting market data
 
  Market_Data market_data;
  market_data.S = 100.;     // $
  market_data.r = 0.0015;     // %
  market_data.sigma = 0.15;  // %
  market_data.T = 1.;       // yr
  market_data.steps = 10;
  market_data.E= 100.;

  Seeds *seed=new Seeds[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];
  Seeds *d_seed;
  initialize(seed, gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID);
  

  hipMalloc((void**)&d_seed, gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID*sizeof(Seeds));
  hipMemcpy(d_seed, seed, gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID*sizeof(Seeds), hipMemcpyHostToDevice);

//dichiaro copie host per cout risultati
/*
Stochastic_Process *SPX= new Stochastic_Process[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];
hipMemcpy(SPX, d_SPX, gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID*sizeof(Stochastic_Process), hipMemcpyDeviceToHost);
*/
double *d_pay;
hipMalloc((void**)&d_pay,  gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID*sizeof(double));
double *pay= new double[ gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];


double *d_stock;
hipMalloc((void**)&d_stock,  gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID*sizeof(double));
double *stock= new double[ gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];

pricing <<<gpu_par.BLOCK_PER_GRID,gpu_par.THREADS_PER_BLOCK>>> (market_data,  gpu_par, par1, par2, par3,d_seed, d_pay,d_stock);

hipMemcpy(pay, d_pay ,  gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID*sizeof(double), hipMemcpyDeviceToHost);

hipMemcpy(stock, d_stock ,  gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID*sizeof(double), hipMemcpyDeviceToHost);


  Rng_Combined *dd_omega = new Rng_Combined[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];
  Stochastic_Process *dd_SPX = new Stochastic_Process[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];
  Opt_European_Call *dd_oec = new Opt_European_Call[gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID];

for (int tid=0; tid< gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID; ++tid){
 dd_omega[tid] = Rng_Combined(seed[tid],par1,par2,par3);
 dd_SPX[tid]= Stochastic_Process(&dd_omega[tid]);
 dd_SPX[tid].Create_Path_Exact(market_data.steps , market_data.S , market_data.T , market_data.sigma , market_data.r );
}

for(int i=0; i<gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID; ++i){
std::cout<< dd_omega[i].Get_Gauss()<< std::endl;
}

/*
for (int i=0; i<  gpu_par.THREADS_PER_BLOCK*gpu_par.BLOCK_PER_GRID; i++){
std::cout<< "pay["<<i<<"]= "<< pay[i];
std::cout<< "   stock["<<i<<"]= "<<stock[i]<< std::endl;
}

std::cout<< "Rng_Combined size= "<< sizeof(Rng_Combined)<<" stoch_process size= "<< sizeof(Stochastic_Process)<<"opt size= "<< sizeof(Opt_European_Call)<< std::endl;
*/


delete[] pay;delete[] stock; delete[] seed;
hipFree(d_pay);hipFree(d_stock); hipFree(d_seed);
  outpath_euler.close(); outpath_exact.close(); out_call.close();
  return 0;
}
