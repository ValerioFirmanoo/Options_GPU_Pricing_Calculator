#include "hip/hip_runtime.h"
#include "opt_european_call.h"
#include <cmath>

// CTOR
__device__ __host__ Opt_European_Call::Opt_European_Call(){
  m_sp = NULL;
 // m_E = 0;
 // m_N = 0;
 // m_S = 0;
//  BlackScholes_Exact();  set option price
}

// CTOR
__device__ __host__ Opt_European_Call::Opt_European_Call(Stochastic_Process *sp){
  m_sp = sp;
 // m_S = sp.Get_path()[sp.Get_steps()-1]; // last element
 // m_E = E;
 // m_N = N;
//  BlackScholes_Exact();  set option price
}

// DTOR
__device__ __host__ Opt_European_Call::~Opt_European_Call(){
  delete[] m_sp;
  m_sp=NULL;
}


// check https://goodcalculators.com/black-scholes-calculator/
/*void Opt_European_Call::BlackScholes_Exact(){
  double d1 = (std::log(m_sp.Get_S()/m_E) + (m_sp.Get_r() + 0.5*m_sp.Get_sigma()*m_sp.Get_sigma())*m_sp.Get_T())/(m_sp.Get_sigma()*std::sqrt(m_sp.Get_T()));   
  double d2 = d1 - m_sp.Get_sigma()*std::sqrt(m_sp.Get_T());
  
  m_F = m_sp.Get_S()*0.5*std::erfc(-d1/std::sqrt(2)) - m_E*0.5*std::erfc(-d2/std::sqrt(2))*std::exp(-m_sp.Get_r()*m_sp.Get_T());
}
*/
