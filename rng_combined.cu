#include "hip/hip_runtime.h"
#include "rng_combined.h"

#include <iostream>
#include <cstdlib>
#include <cmath>

// CTOR
 __device__ __host__ Rng_Combined::Rng_Combined() {
  m_rand = 0;
  m_seeds.sa = 0; m_seeds.sb = 0; m_seeds.sc = 0; m_seeds.sd = 0;
  m_p1.k1 = 0; m_p1.k2 = 0; m_p1.k3 = 0; m_p1.m = 0;
  m_p2.k1 = 0; m_p2.k2 = 0; m_p2.k3 = 0; m_p2.m = 0;
  m_p3.k1 = 0; m_p3.k2 = 0; m_p3.k3 = 0; m_p3.m = 0;
  m_rand_bm1 = 0; m_rand_bm2 = 0;
}

// Value CTOR
 __device__ __host__ Rng_Combined::Rng_Combined(Seeds seeds, Tausworthe_Parameters p1, Tausworthe_Parameters p2, Tausworthe_Parameters p3) {
  m_rand = 0;
  m_seeds = seeds;
  m_p1 = p1; m_p2 = p2; m_p3 = p3;
  m_a=0; //initializzo variabile ausiliaria

  m_tau1 = Rng_Tausworthe(m_seeds.sa, m_p1);
  m_tau2 = Rng_Tausworthe(m_seeds.sb, m_p2);
  m_tau3 = Rng_Tausworthe(m_seeds.sc, m_p3);
  m_lcg = Rng_Lcg(m_seeds.sd);
}

 __device__ __host__ void Rng_Combined::Random() {
  m_tau1.Random();
  m_tau2.Random();
  m_tau3.Random();
  m_lcg.Random();
  
  m_seeds.sa = m_tau1.Get_Seed();
  m_seeds.sb = m_tau2.Get_Seed();
  m_seeds.sc = m_tau3.Get_Seed();
  m_seeds.sd = m_lcg.Get_Seed();

  m_rand = 2.3283064365387e-10*(double)((m_seeds.sa)^(m_seeds.sb)^(m_seeds.sc)^(m_seeds.sd));
}

 __device__ __host__ void Rng_Combined::RandomBoxMuller(){
  m_tau1.Random();
  m_tau2.Random();
  m_tau3.Random();
  m_lcg.Random();
  m_seeds.sa = m_tau1.Get_Seed();
  m_seeds.sb = m_tau2.Get_Seed();
  m_seeds.sc = m_tau3.Get_Seed();
  m_seeds.sd = m_lcg.Get_Seed();
  double g1 = 2.3283064365387e-10*(double)((m_seeds.sa)^(m_seeds.sb)^(m_seeds.sc)^(m_seeds.sd));

  m_tau1.Random();
  m_tau2.Random();
  m_tau3.Random();
  m_lcg.Random();
  m_seeds.sa = m_tau1.Get_Seed();
  m_seeds.sb = m_tau2.Get_Seed();
  m_seeds.sc = m_tau3.Get_Seed();
  m_seeds.sd = m_lcg.Get_Seed();
  double g2 = 2.3283064365387e-10*(double)((m_seeds.sa)^(m_seeds.sb)^(m_seeds.sc)^(m_seeds.sd));
  
  m_rand_bm1 = std::sqrt(-2.*std::log(g1))*std::cos(2.*M_PI*g2);  
  m_rand_bm2 = std::sqrt(-2.*std::log(g1))*std::sin(2.*M_PI*g2);  
}


 __device__ __host__ double Rng_Combined::Get_Gauss(){
  m_a=m_rand_bm1;
  m_rand_bm1=m_rand_bm2;
  return m_a;
} 
