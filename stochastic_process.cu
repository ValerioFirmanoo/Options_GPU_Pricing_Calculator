#include "hip/hip_runtime.h"
#include "stochastic_process.h"

__device__ __host__ Stochastic_Process::Stochastic_Process() {
//  m_md.S = 0.;
//  m_md.r = 0.;
//  m_md.sigma = 0.;
//  m_md.T = 0.;
//  m_md.steps = 0;
  m_omega = NULL;
  m_path = NULL;
  //m_S_aux = 0.;
  //m_S_aux = m_md.S;
}

__device__ __host__ Stochastic_Process::Stochastic_Process(Rng_Combined *omega) {
  //m_md = md;
  m_omega = omega;
  m_path = NULL;
  //m_S_aux = 0.;
  //m_S_aux = md.S; // m_S_aux = m_md.S
}

/*
__device__ __host__ Stochastic_Process::Stochastic_Process(const Stochastic_Process &copy){
//  std::cout << "copy ctor" << std::endl;
  m_S_aux = copy.Get_S_aux();
  m_md = copy.Get_Market_Data(); // struct memberwise copy
  m_omega = copy.Get_omega();
  m_path = new double[m_md.steps]; 
  for (int i=0; i<copy.Get_Market_Data().steps; ++i){
    m_path[i] = copy.Get_path()[i];
  }
}
*/

 __device__ __host__ Stochastic_Process::~Stochastic_Process(){
  delete[] m_path;
  m_path = NULL;
}


/*

__device__ __host__ void Stochastic_Process::Integrate_Euler() {
  double dT = m_md.T/(1.*m_md.steps);
  m_omega.RandomBoxMuller();
  m_md.S = m_md.S*(1 + m_md.r*dT + m_md.sigma*std::sqrt(dT)*m_omega.Get_Gauss1()); 
  m_S_aux = m_md.S*(1 + m_md.r*dT + m_md.sigma*std::sqrt(dT)*m_omega.Get_Gauss2()); 
}

__device__ __host__ void Stochastic_Process::Integrate_Exact() {
  double dT = m_md.T/(1.*m_md.steps);
  m_omega.RandomBoxMuller();
  m_md.S = m_md.S*std::exp((m_md.r-0.5*m_md.sigma*m_md.sigma)*dT + m_md.sigma*std::sqrt(dT)*m_omega.Get_Gauss1());
  m_S_aux = m_md.S*std::exp((m_md.r-0.5*m_md.sigma*m_md.sigma)*dT + m_md.sigma*std::sqrt(dT)*m_omega.Get_Gauss2());
}

__device__ __host__ void Stochastic_Process::Create_Path_Euler() {
  for (int i=0; i<m_md.steps; i+=2){
    Integrate_Euler();
    m_path[i] = m_md.S;
    m_path[i+1] = m_S_aux;
    m_md.S = m_S_aux;
  }
}

__device__ __host__ void Stochastic_Process::Create_Path_Exact() {
  for (int i=0; i<m_md.steps; i+=2){
    Integrate_Exact();
    m_path[i] = m_md.S;
    m_path[i+1] = m_S_aux;
    m_md.S = m_S_aux;
  }
}
*/

__device__ __host__ void Stochastic_Process::Integrate_Euler(int nsteps, double S, double aux , double dT, double sigma, double r) {
  m_omega->RandomBoxMuller();
  S = S*(1. + r*dT + sigma*std::sqrt(dT)*m_omega->Get_Gauss()); 
  aux = S*(1. + r*dT + sigma*std::sqrt(dT)*m_omega->Get_Gauss()); 
}


__device__ __host__ void Stochastic_Process::Integrate_Exact(int nsteps, double S, double aux , double dT, double sigma, double r) {
  m_omega->RandomBoxMuller();
  S = S*std::exp((r-0.5*sigma*sigma)*dT + sigma*std::sqrt(dT)*m_omega->Get_Gauss());
  aux = S*std::exp((r-0.5*sigma*sigma)*dT + sigma*std::sqrt(dT)*m_omega->Get_Gauss());
}


__device__ __host__ void Stochastic_Process::Create_Path_Euler(int nsteps, double S0, double T, double sigma, double r ) {
  m_path= new double [nsteps];
  double S=S0;
  double aux=0;
  double dT = T/(1.*nsteps);
  for (int i=0; i<nsteps; i+=2){
    Integrate_Euler(nsteps , S , aux , dT , sigma , r);
    m_path[i] = S;
    m_path[i+1] = aux;
    S = aux;
  }
}

__device__ __host__ void Stochastic_Process::Create_Path_Exact(int nsteps, double S0 , double T, double sigma, double r) {
  m_path= new double [nsteps];
  double S=S0;
  double aux=0;
  double dT = T/(1.*nsteps);
  for (int i=0; i<nsteps; i+=2){
    Integrate_Exact(nsteps , S , aux , dT , sigma , r);
    m_path[i] = S;
    m_path[i+1] = aux;
    S = aux;
  }
}
